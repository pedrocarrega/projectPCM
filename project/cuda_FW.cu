#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime_api.h>

#include <math.h>
#include <sys/time.h>

#ifndef __CUDACC__ 
#define __CUDACC__
#endif
#include <hip/device_functions.h>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define GRAPH_SIZE 2048
#define WORK_SIZE 96
#define NTHREADS 32
#define BLOCKS 3

#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)

#define INF 0x1fffffff


//createGraph
void generate_random_graph(int* output, int graph_size) {
	int i, j;

	srand(0xdadadada);

	for (i = 0; i < graph_size; i++) {
		for (j = 0; j < graph_size; j++) {
			if (i == j) {
				D(i, j) = 0;
			}
			else {
				int r;
				r = rand() % 40;
				if (r > 20) {
					r = INF;
				}

				D(i, j) = r;
			}
		}
	}
}

//calcOnePositionPerThread e deixar o schedualing para a gpu, fazendo assim todas as posicoes da matriz
__global__ void calcOnePosPerThread(int* output, int graph_size, int k)
{
	int i = (blockIdx.x * blockDim.x + threadIdx.x);
	int j = (blockIdx.y * blockDim.y + threadIdx.y);

	//while (i < graph_size && j < graph_size) {
		if (D(i, k) + D(k, j) < D(i, j)) {
			D(i, j) = D(i, k) + D(k, j);
		}
		//i += blockDim.x * gridDim.x;
		//j += blockDim.y * gridDim.y;
	//}
}

__global__ void calThreadPerColumn(int* output, const int graph_size, int numThreads, int k)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int localStorageRegister;


	while (i < graph_size) {

		localStorageRegister = D(i, k);

		for (int j = 0; j < graph_size; j++)
		{
			if (localStorageRegister + D(k, j) < D(i, j)) {
				D(i, j) = localStorageRegister + D(k, j);
			}
		}
		i = i + numThreads;
	}
}

__device__ int barrier = 0;

/*
Problemas em todos os que usam atomic possivelmente devido a estar a calcular mal o num max
de threads/blocks/warps que se pode ter na totalidade assim como por SM, ver descriao nas doubts.txt
*/
__global__ void calcWithAtomic(int* output, int graph_size)
{
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * WORK_SIZE;
	int j = (blockIdx.y * blockDim.y + threadIdx.y) * WORK_SIZE;
	int k = 0;
	int numBlocks = gridDim.x * gridDim.y;

	while (k < graph_size) {
		if (threadIdx.x == 0 && threadIdx.y == 0)
		printf("Before\n");
		for (int x = i; x < i + WORK_SIZE; x++)
		{
			for (int y = j; y < j + WORK_SIZE; y++)
			{
				if (x < graph_size && y < graph_size) {
					if (D(x, k) + D(k, y) < D(x, y)) {
						D(x, y) = D(x, k) + D(k, y);
					}
				}else{
					break;
				}
			}
			if (threadIdx.x == 0 && threadIdx.y == 0)
			printf("AFTER: %d\n", x);
		}

		k++;
		/*if (threadIdx.x == 0 && threadIdx.y == 0)
		printf("After\n");
		*/
		//bloco perde-se
		if (threadIdx.x == 0 && threadIdx.y == 0) {
			atomicAdd(&barrier,1);
			printf("barrier: %d\n", barrier);
			while (barrier % numBlocks != 0);
		}
		
		__syncthreads();

	}
}

__global__ void calcWithoutAtomic(int* output, int graph_size, int k, int workPerThread)
{
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * workPerThread;
	int j = (blockIdx.y * blockDim.y + threadIdx.y) * workPerThread;

	//int xk, ky;

	for (int x = i; x < i + workPerThread; x++)
	{
		//xk = D(x, k);
		for (int y = j; y < j + workPerThread; y++)
		{
			//ky = D(k, y);
			if (x < graph_size && y < graph_size) {
				if (D(x, k) + D(k, y) < D(x, y)) {
					D(x, y) = D(x, k) + D(k, y);
				}
			}
		}
	}
}

/*
__global__ void calcSharedWithAtomic(int* output, int graph_size)
{
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * WORK_SIZE;
	int j = (blockIdx.y * blockDim.y + threadIdx.y) * WORK_SIZE;
	int k = 0;
	int numBlocks = gridDim.x * gridDim.y;
	
	__shared__ int valuesX[NTHREADS][NTHREADS];
	__shared__ int valuesY[NTHREADS][NTHREADS][WORK_SIZE];

	while (k < graph_size) {
		for (int x = i; x < i + WORK_SIZE; x++)
		{
			if (x < graph_size) {
				valuesX[threadIdx.x][threadIdx.y] = D(x, k);
			}
			for (int y = j; y < j + WORK_SIZE; y++)
			{
				if (x < graph_size && y < graph_size) {
					if (x == i) {
						valuesY[threadIdx.x][threadIdx.y][y-j] = D(k, y);
					}
					if (valuesX[threadIdx.x][threadIdx.y] + valuesY[threadIdx.x][threadIdx.y][y - j] < D(x, y)) {
						D(x, y) = valuesX[threadIdx.x][threadIdx.y] + valuesY[threadIdx.x][threadIdx.y][y - j];
					}
				}
			}
		}

		k++;

		if (threadIdx.x == 0 && threadIdx.y) {
			atomicAdd(&barrier,1);
			while (barrier % numBlocks != 0);
		}
		__syncthreads();

	}
}


__global__ void calcSharedWithoutAtomic(int* output, int graph_size, int k, int workPerThread)
{
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * workPerThread;
	int j = (blockIdx.y * blockDim.y + threadIdx.y) * workPerThread;
	//printf("xt = %d | yt = %d\n", threadIdx.x, threadIdx.y);
	int xT = threadIdx.x;
	int yT = threadIdx.y;

	__shared__ int valuesX[NTHREADS][NTHREADS];
	__shared__ int valuesY[NTHREADS*NTHREADS][WORK_SIZE];

	int currT = xT * blockDim.x + yT;

	//printf("workPT %d , blockdim = %d ", workPerThread, blockDim.x);
	/*
	Por enquanto o valuesX faz ser um pouco mais rapido mas o valuesY faz ficar bastante mais lento, implementar com shared memory
	*//*
	for (int x = i; x < i + workPerThread; x++)
	{
		
		if (x < graph_size) {
			valuesX[xT][yT] = D(x, k);
		}/*
		if (threadIdx.x == 0 && threadIdx.y == 0) {
			array[blockDim.x * blockIdx.x] = D(x, k);
		}
		__syncthreads;*//*
		for (int y = j; y < j + workPerThread; y++)
		{
			//values[threadX][threadY] = D(x, k);
			//ky = D(k, y);
			if (x < graph_size && y < graph_size) {
				if (x == i) {
					valuesY[currT][y - j] = D(k, y);
				}
				if (valuesX[xT][yT] + valuesY[currT][y - j] < D(x, y)) {
					D(x, y) = valuesX[xT][yT] + valuesY[currT][y - j];
				}
			}
		}
	}
}

/*
__global__ void sharedCalcWithoutAtomic(int* output, int graph_size, int k, const int workPerThread)
{
	//size_t workSize = (workPerThread * blockDim.x)^2;

	//const int test = blockDim.x;
	
	__shared__ int array[10 * 11];
	
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * workPerThread;
	int j = (blockIdx.y * blockDim.y + threadIdx.y) * workPerThread;

	for (int x = i; x < i + workPerThread; x++)
	{
		if (threadIdx.x == 0 && threadIdx.y == 0) {
			array[blockDim.x * blockIdx.x] = D(x, k);
		}
		__syncthreads;
		for (int y = j; y < j + workPerThread; y++)
		{
			if (x < graph_size && y < graph_size) {
				if (D(x, k) + D(k, y) < D(x, y)) {
					D(x, y) = D(x, k) + D(k, y);
				}
			}
		}
	}
}
*/


//sequencial GPU
__global__ void calculateSequencialGPU(int* output, int graph_size)
{
	int i, j, k;

	for (k = 0; k < graph_size; k++) {
		for (i = 0; i < graph_size; i++) {
			for (j = 0; j < graph_size; j++) {
				if (D(i, k) + D(k, j) < D(i, j)) {
					D(i, j) = D(i, k) + D(k, j);
				}
			}
		}
	}
}

void floyd_warshall_gpu(const int* graph, int graph_size, int* output) {

	int* dev_a;
	hipMalloc(&dev_a, sizeof(int) * graph_size * graph_size);
	hipMemcpy(dev_a, graph, sizeof(int) * graph_size * graph_size, hipMemcpyHostToDevice);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	int NThreads = NTHREADS;
	//for safety
	if (NThreads > 32) {
		NThreads = sqrt(prop.maxThreadsPerBlock);
	}

	
	//int maxMemSize = prop.sharedMemPerBlock;
	//int maxBlocksPerAxis = sqrt(prop.multiProcessorCount * (prop.maxThreadsPerMultiProcessor / (NThreads * NThreads)));
	/*
	int maxThreadsPerAxis = maxBlocksPerAxis * NThreads;
	int workPerThread = ((graph_size) / maxThreadsPerAxis) + 1;

	fprintf(stderr, "work %d\nthreads %d\n", workPerThread, NThreads);
	*/

	//dim3 threads(NThreads, NThreads);
	//dim3 blocks(maxBlocksPerAxis, maxBlocksPerAxis);

	//printf("blockPerAxis = %d\n", maxBlocksPerAxis);

	//int t = 64;
	//int b = prop.multiProcessorCount * (prop.maxThreadsPerMultiProcessor / t);
	
	//calculateSequencialGPU << <1, 1 >> > (dev_a, graph_size);
/*
	int blocks;
	int threads;
	cudaOccupancyMaxPotentialBlockSize (&blocks, &threads, calcWithAtomic, 0, GRAPH_SIZE*GRAPH_SIZE);
	blocks = sqrt(blocks);
	threads = sqrt(threads);
	printf("workPerThread to be defined as %d\n", threads*blocks);
	
/*	
	if(threads % 2 != 0){
		threads++;
	}
*/
	
/*
	if(blocks % 2 != 0){
		blocks++;
	}*/


	
	/*
	for (int k = 0; k < graph_size; k++) {
		//calcSharedWithoutAtomic <<<blocks, threads>>> (dev_a, graph_size, k);
		//calcWithoutAtomic <<<blocks, threads>>> (dev_a, graph_size, k);
		calcOnePosPerThread <<<dim3(GRAPH_SIZE/8,GRAPH_SIZE/8), dim3(8,8)>>>(dev_a, graph_size,k);
		//calThreadPerColumn <<<b, t >>> (dev_a, graph_size, t * b, k);
	}
	*/
	
	
	
	//fprintf(stderr, "blocks: %d\nthreads: %d\n", blocks, threads);
	
	calcWithAtomic <<<dim3(BLOCKS,BLOCKS), dim3(NTHREADS,NTHREADS)>>> (dev_a, graph_size);
	//calcSharedWithAtomic <<<blocks, threads >>> (dev_a, graph_size);
	//calcSIMDSharedWithAtomic <<<blocks, threads >>> (dev_a, graph_size);

	hipError_t err = hipMemcpy(output, dev_a, sizeof(int) * graph_size * graph_size, hipMemcpyDeviceToHost);
	gpuErrchk(err);
	hipFree(dev_a);
}

void floyd_warshall_cpu(const int* graph, int graph_size, int* output) {
	int i, j, k;

	memcpy(output, graph, sizeof(int) * graph_size * graph_size);

	for (k = 0; k < graph_size; k++) {
		for (i = 0; i < graph_size; i++) {
			for (j = 0; j < graph_size; j++) {
				if (D(i, k) + D(k, j) < D(i, j)) {
					D(i, j) = D(i, k) + D(k, j);
				}
			}
		}
	}
}

int main(int argc, char** argv) {

	#define TIMER_START() gettimeofday(&tv1, NULL)
	#define TIMER_STOP()                                                           \
  		gettimeofday(&tv2, NULL);                                                    \
  		timersub(&tv2, &tv1, &tv);                                                   \
  		time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0

  struct timeval tv1, tv2, tv;
  float time_delta;	
 
  int* graph, * output_cpu, * output_gpu;
	int size;

	size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;

	graph = (int*)malloc(size);
	assert(graph);

	output_cpu = (int*)malloc(size);
	assert(output_cpu);
	memset(output_cpu, 0, size);

	output_gpu = (int*)malloc(size);
	assert(output_gpu);

	generate_random_graph(graph, GRAPH_SIZE);

	fprintf(stderr, "running on cpu...\n");
	TIMER_START();
	//floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
	TIMER_STOP();
	fprintf(stderr, "%f seconds\n", time_delta);

	fprintf(stderr, "running on gpu...\n");
	TIMER_START();
	floyd_warshall_gpu(graph, GRAPH_SIZE, output_gpu);
	TIMER_STOP();
	fprintf(stderr, "%f seconds\n", time_delta);



	if (memcmp(output_cpu, output_gpu, size) != 0) {
		fprintf(stderr, "FAIL!\n");
	}
	else {
		fprintf(stderr, "Verified!\n");
	}

	return 0;
}
